#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#ifdef __linux__
#include <sys/time.h>
#endif

#include <stdlib.h> 
#include <cmath>

#define Double double
#define N_STREAMS 4

// When comparing the output between CPU and GPU implementation, 
// the precision of the floating-point operations might differ between different versions, 
// which can translate into srounding error differences. 
// Hence, use a margin error range when comparing both versions.

// Please implement a simple vectorAdd program that sums two vectors and stores the results into a third vector. 
// You will understand how to index 1D arrays inside a GPU kernel. 
// Please complete the following main steps in your code. You can create your own code, or, 
// use the following code template (Download Code Template Here hw2_ex1_template.cu 
// Download hw2_ex1_template.cu ) and edit code parts demarcated by the //@@ comment lines. 

__device__ Double addNum(Double x1, Double x2) {
    return x1 + x2;
}

__global__ void vecAdd(Double *in1, Double *in2, Double *out, int len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < len) {
    const Double x = addNum(in1[i], in2[i]);
    out[i] = x;
  }
}

//@@ Insert code to implement timer start

long int cpuSecond() {
    #ifdef __linux__
    struct timeval timer;
    gettimeofday(&timer, NULL);
    return timer.tv_sec * 1000000 + timer.tv_usec;
    #endif

    return 0;
}

//----------
int main(int argc, char **argv) {
int input_length;
int S_seg;
long int iStart;
long int iElaps;
Double *hostInput1;
Double *hostInput2;
Double *hostOutput;
Double *resultRef;
Double *deviceInput1;
Double *deviceInput2;
Double *deviceOutput;

//@@ Insert code below to read in input_length from args
if (argc != 3) {
    printf("!!\tERROR: Expected 2 arguments, got %d\n", argc - 1);
    exit(1);
}

printf("->\tStart of execution...\n");

input_length = atoi(argv[1]);                           //ASCII to integer
S_seg  = atoi(argv[2]);    

printf("The input length is %d\n", input_length);
printf("Seg: %d\n", S_seg);

// Allocate Host memory for input and output
hipHostAlloc(&hostInput1, input_length * sizeof(Double),hipHostMallocDefault);
hipHostAlloc(&hostInput2, input_length * sizeof(Double),hipHostMallocDefault);
hipHostAlloc(&hostOutput, input_length * sizeof(Double),hipHostMallocDefault);
resultRef  = (Double*)malloc(input_length * sizeof(Double));

//@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  for (int i = 0; i < input_length; i++) {
    hostInput1[i] = rand()/(Double)RAND_MAX;
    hostInput2[i] = rand()/(Double)RAND_MAX;
    resultRef[i]  = hostInput1[i] + hostInput2[i];
  }

printf("->\tRandom input vectors created...\n");

//@@ Insert code below to allocate GPU memory here
hipMalloc(&deviceInput1, input_length * sizeof(Double));
hipMalloc(&deviceInput2, input_length * sizeof(Double));
hipMalloc(&deviceOutput, input_length * sizeof(Double));

printf("->\tReference vector created...\n");
//@@ Insert code to below to Copy memory to the GPU here
iStart = cpuSecond();

// create stream
hipStream_t stream[N_STREAMS];

for (int i = 0; i < N_STREAMS; ++i)
    hipStreamCreate(&stream[i]); 

int number_of_segments = (input_length + S_seg - 1) / S_seg;
int segment_size = (input_length + number_of_segments - 1) / number_of_segments;

for (int i = 0; i < number_of_segments; i++) {
    int offset = i * segment_size;
    int length = min(S_seg, input_length - offset);

    hipStream_t t_stream = stream[i % N_STREAMS];

    //@@ Initialize the 1D grid and block dimensions here
    dim3 dimBlock(1024, 1, 1);
    dim3 dimGrid(length / 1024 + (length % 1024 != 0), 1, 1);

    hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], length * sizeof(Double), hipMemcpyHostToDevice,t_stream);
    hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], length * sizeof(Double), hipMemcpyHostToDevice,t_stream); 
    vecAdd<<<dimGrid, dimBlock,0,t_stream>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], length);
    hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], length * sizeof(Double), hipMemcpyDeviceToHost,t_stream);
}

hipDeviceSynchronize();
iElaps = cpuSecond() - iStart;
printf("~~\tTime to execute the H2D+kernel+D2H: %ld\n", iElaps);

//@@ Insert code below to compare the output with the reference
for (int i = 0; i < input_length; i++){
    if(abs(resultRef[i] - hostOutput[i]) >= 1){
        printf("!!\tError in position (%d);\n", i);
        printf("!!\tExpected %f, found %f\n\n", resultRef[i], hostOutput[i]);
        break;
    }
}

for (int i = 0; i < N_STREAMS; ++i)
    hipStreamDestroy( stream[i] );
//@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

//@@ Free the CPU memory here
  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);
  free(resultRef);

  return 0;
}